
#include <hip/hip_runtime.h>
struct DummyComplexFloat
{
	public: float Real;
	public: float Imag;

	// Methods
	__device__ DummyComplexFloat(float  r, float  i)
	{
		Real = r;
		Imag = i;
	}


	__device__ DummyComplexFloat  Add(DummyComplexFloat  c)
	{
		return DummyComplexFloat((Real + c.Real), (Imag + c.Imag));
	}
};